#include "hip/hip_runtime.h"
#include <math.h>
#include <limits>
#include <iostream>

//GPU Add
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// for timing
#include <chrono>
#include <ctime>
// user include

#include "CLUEAlgoGPU.h"


__global__ void kernel_compute_histogram( LayerTilesGPU *d_hist, 
                                          PointsPtr d_points, 
                                          int numberOfPoints
                                          )
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < numberOfPoints) {
    // push index of points into tiles
    d_hist[d_points.layer[i]].fill(d_points.x[i], d_points.y[i], i);
  }
} //kernel


__global__ void kernel_compute_density( LayerTilesGPU *d_hist, 
                                        PointsPtr d_points, 
                                        float dc,
                                        int numberOfPoints
                                        ) 
{ 
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numberOfPoints){
    double rhoi{0.};
    int layeri = d_points.layer[i];
    float xi = d_points.x[i];
    float yi = d_points.y[i];

    // get search box 
    int4 search_box = d_hist[layeri].searchBox(xi-dc, xi+dc, yi-dc, yi+dc);

    // loop over bins in the search box
    for(int xBin = search_box.x; xBin < search_box.y+1; ++xBin) {
      for(int yBin = search_box.z; yBin < search_box.w+1; ++yBin) {

        // get the id of this bin
        int binId = d_hist[layeri].getGlobalBinByBin(xBin,yBin);
        // get the size of this bin
        int binSize  = d_hist[layeri][binId].size();

        // interate inside this bin
        for (int binIter = 0; binIter < binSize; binIter++) {
          int j = d_hist[layeri][binId][binIter];
          // query N_{dc_}(i)
          float xj = d_points.x[j];
          float yj = d_points.y[j];
          float dist_ij = std::sqrt((xi-xj)*(xi-xj) + (yi-yj)*(yi-yj));
          if(dist_ij < dc) { 
            // sum weights within N_{dc_}(i)
            rhoi += d_points.weight[j];              
          }
        } // end of interate inside this bin
      }
    } // end of loop over bins in search box
    d_points.rho[i] = rhoi;
  }
} //kernel


__global__ void kernel_compute_distanceToHigher(LayerTilesGPU* d_hist, 
                                                PointsPtr d_points, 
                                                float d0,
                                                int numberOfPoints
                                                ) 
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < numberOfPoints){
    int layeri = d_points.layer[i];

    float deltai = std::numeric_limits<float>::max();
    int nearestHigheri = -1;
    float xi = d_points.x[i];
    float yi = d_points.y[i];
    float rhoi = d_points.rho[i];

    // get search box 
    int4 search_box = d_hist[layeri].searchBox(xi-d0, xi+d0, yi-d0, yi+d0);

    // loop over all bins in the search box
    for(int xBin = search_box.x; xBin < search_box.y+1; ++xBin) {
      for(int yBin = search_box.z; yBin < search_box.w+1; ++yBin) {
        // get the id of this bin
        int binId = d_hist[layeri].getGlobalBinByBin(xBin,yBin);
        // get the size of this bin
        int binSize  = d_hist[layeri][binId].size();

        // interate inside this bin
        for (int binIter = 0; binIter < binSize; binIter++) {
          int j = d_hist[layeri][binId][binIter];
          // query N'_{dc_}(i)
          float xj = d_points.x[j];
          float yj = d_points.y[j];
          float dist_ij = std::sqrt((xi-xj)*(xi-xj) + (yi-yj)*(yi-yj));
          bool foundHigher = (d_points.rho[j] > rhoi);
          // in the rare case where rho is the same, use detid
          foundHigher = foundHigher || ( (d_points.rho[j] == rhoi) && (j>i));
          if(foundHigher && dist_ij < d0) { // definition of N'_{dc_}(i)
            // find the nearest point within N'_{dc_}(i)
            if (dist_ij<deltai) {
              // update deltai and nearestHigheri
              deltai = dist_ij;
              nearestHigheri = j;
            }
          }
        } // end of interate inside this bin
      }
    } // end of loop over bins in search box
    d_points.delta[i] = deltai;
    d_points.nearestHigher[i] = nearestHigheri;
  }
} //kernel



__global__ void kernel_find_clusters( GPU::VecArray<int,maxNSeeds>* d_seeds,
                                      GPU::VecArray<int,maxNFollowers>* d_followers,
                                      PointsPtr d_points,
                                      float dc, float d0, float rhoc,
                                      int numberOfPoints
                                      ) 
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i < numberOfPoints) {
    // initialize clusterIndex
    d_points.clusterIndex[i] = -1;
    // determine seed or outlier
    float deltai = d_points.delta[i];
    float rhoi = d_points.rho[i];
    bool isSeed = (deltai > dc) && (rhoi >= rhoc);
    bool isOutlier = (deltai > d0) && (rhoi < rhoc);

    if (isSeed) {
      // set isSeed as 1
      d_points.isSeed[i] = 1;
      d_seeds[0].push_back(i); // head of d_seeds
    } else {
      if (!isOutlier) {
        // register as follower at its nearest higher
        d_followers[d_points.nearestHigher[i]].push_back(i);  
      }
    }
  }
} //kernel


__global__ void kernel_assign_clusters( GPU::VecArray<int,maxNSeeds>* d_seeds, 
                                        GPU::VecArray<int,maxNFollowers>* d_followers,
                                        PointsPtr d_points
                                        )
{

  int idxCls = blockIdx.x * blockDim.x + threadIdx.x;

  if (idxCls < d_seeds[0].size()){

    int localStack[localStackSizePerSeed] = {-1};
    int localStackSize = 0;

    // asgine cluster to seed[idxCls]
    int idxThisSeed = d_seeds[0][idxCls];
    d_points.clusterIndex[idxThisSeed] = idxCls;
    // push_back idThisSeed to localStack
    localStack[localStackSize] = idxThisSeed;
    localStackSize++;

    // process all elements in localStack
    while (localStackSize>0){
      // get last element of localStack
      int idxEndOflocalStack = localStack[localStackSize-1];

      int temp_clusterIndex = d_points.clusterIndex[idxEndOflocalStack];
      GPU::VecArray<int,maxNFollowers> temp_followers = d_followers[idxEndOflocalStack];
              
      // pop_back last element of localStack
      localStack[localStackSize-1] = -1;
      localStackSize--;

      // loop over followers of last element of localStack
      for( int j : temp_followers){
        // pass id to follower
        d_points.clusterIndex[j] = temp_clusterIndex;
        // push_back follower to localStack
        localStack[localStackSize] = j;
        localStackSize++;
      }
    }
  }
} //kernel






void CLUEAlgoGPU::makeClusters( ) {

  copy_todevice();
  clear_set();

  ////////////////////////////////////////////
  // calcualte rho, delta and find seeds 
  // 1 point per thread
  ////////////////////////////////////////////
  const dim3 blockSize(1024,1,1);
  const dim3 gridSize(ceil(points_.n/1024.0),1,1);
  kernel_compute_histogram <<<gridSize,blockSize>>>(d_hist, d_points, points_.n);
  kernel_compute_density <<<gridSize,blockSize>>>(d_hist, d_points, dc_, points_.n);
  kernel_compute_distanceToHigher <<<gridSize,blockSize>>>(d_hist, d_points, d0_, points_.n);
  kernel_find_clusters <<<gridSize,blockSize>>>(d_seeds, d_followers, d_points, dc_,d0_,rhoc_, points_.n);  
  
  ////////////////////////////////////////////
  // assign clusters
  // 1 point per seeds
  ////////////////////////////////////////////
  const dim3 gridSize_nseeds(ceil(maxNSeeds/1024.0),1,1);
  kernel_assign_clusters <<<gridSize_nseeds,blockSize>>>(d_seeds, d_followers, d_points);

  copy_tohost();
}
